#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/scan.h>
#include <pthread.h>
#include "hip/hip_runtime.h"

#include <stdint.h>
#include <stdio.h>
#include <tgmath.h>
#include <sys/time.h>
#include <assert.h>

#include "half.hpp"

#include "plotecg.h"
#include "kernels.h"

#define checkCuda(result) _checkCuda(result, __LINE__, __FILE__)
#define KERNEL(func) func<<<num_blocks, threads_per_block>>>

inline
hipError_t _checkCuda(hipError_t result, int l, const char * f)
{
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s (%s: line %d)\n", hipGetErrorString(result), f, l);
    assert(result == hipSuccess);
  }
  return result;
}

double get_time(void) {
  struct timeval t;

  gettimeofday(&t, NULL);
  return (double)t.tv_sec*1000000.0 + ((double)t.tv_usec);
}

inline void to_fp16(uint16_t * out, float * in, int len) {
  int i;
  for (i = 0; i < len; i++) {
    out[i] = half_float::detail::float2half<std::round_indeterminate>(in[i]);
  }
}

inline void turning_point_compress(float * output, float * input, int input_len)
{
  int idx;
  int output_len = input_len / 2;
  output[0] = input[0];
  for (idx = 1; idx < output_len; idx++) {
    if ((input[2*idx]-output[idx-1])*(input[2*idx+1]-input[2*idx]) < 0) {
      output[idx] = input[2*idx];
    } else {
      output[idx] = input[2*idx+1];
    }
  }
}

struct tp_arg {
  uint16_t * output;
  float * input;
  float * inter;
  float * cinter;
  int len;
};

void * tp_worker(void * _args) {
  struct tp_arg * args = (struct tp_arg *) _args;
  uint16_t * output = args -> output;
  float * input = args -> input;
  float * inter = args -> inter;
  float * cinter = args -> cinter;
  int len = args -> len;
  turning_point_compress(inter, input, len);
  turning_point_compress(cinter, inter, len / 2);
  to_fp16(output, cinter, len / 4);
  pthread_exit(NULL);
}

void parallel_turning_point_compress(uint16_t * output,
                                     float * input,
                                     int input_len)
{
  int num_threads = 8;
  int tid;
  struct tp_arg thread_args[num_threads];
  pthread_t threads[num_threads];
  pthread_attr_t th_attr;
  pthread_attr_init(&th_attr);
  pthread_attr_setdetachstate(&th_attr, PTHREAD_CREATE_JOINABLE);
  int chunk_size = input_len / num_threads;
  float * inter = (float *) malloc((input_len * sizeof(float)) / 2);
  float * cinter = (float *) malloc((input_len * sizeof(float)) / 4);
  assert(inter);
  assert(cinter);
  for (tid = 0; tid < num_threads; tid++) {
    (&thread_args[tid]) -> output = & output[(chunk_size * tid) / 4];
    (&thread_args[tid]) -> cinter = & cinter[(chunk_size * tid) / 4];
    (&thread_args[tid]) -> inter = & inter[(chunk_size * tid) / 2];
    (&thread_args[tid]) -> input = & input[chunk_size * tid];
    (&thread_args[tid]) -> len = chunk_size;
    pthread_create(&threads[tid], &th_attr, tp_worker, (void *) & thread_args[tid]);
  }
  for (tid = 0; tid < num_threads; tid++) {
    pthread_join(threads[tid], NULL);
  }
  pthread_attr_destroy(&th_attr);
  free(inter);
  free(cinter);
}

void inclusive_scan(int * out, int * in, int len) {
  thrust::device_ptr<int> in_p = thrust::device_pointer_cast(in);
  thrust::device_ptr<int> out_p = thrust::device_pointer_cast(out);
  thrust::inclusive_scan(in_p, in_p+len, out_p);
}

void exclusive_scan(int * out, int * in, int len) {
  thrust::device_ptr<int> in_p = thrust::device_pointer_cast(in);
  thrust::device_ptr<int> out_p = thrust::device_pointer_cast(out);
  thrust::exclusive_scan(in_p, in_p+len, out_p);
}

void device_index(int * ary, int * last_val, int idx) {
  hipMemcpy(last_val, & ary[idx], sizeof(int), hipMemcpyDeviceToHost);
}

void synchronize_and_merge(int ** merged_out,
                           int * merged_length_out,
                           int * d_lead1,
                           int * d_lead2,
                           int * d_lead3,
                           int lead_length,
                           int chunk_length)
{
  int * lead1, * lead2, * lead3;
  size_t chunk_size = chunk_length * sizeof(int);
  int start1 = 0, start2 = 0, start3 = 0;
  int offset1, offset2, offset3;
  int minstart, maxstart;
  int i;
  int sync_length;
  int threads_per_block = 256;
  int num_blocks;
  // Allocate small chunks
  lead1 = (int *) malloc(chunk_size);
  assert(lead1);
  lead2 = (int *) malloc(chunk_size);
  assert(lead2);
  lead3 = (int *) malloc(chunk_size);
  assert(lead3);
  // Copy back
  checkCuda( hipMemcpy(lead1, d_lead1, chunk_size, hipMemcpyDeviceToHost) );
  checkCuda( hipMemcpy(lead2, d_lead2, chunk_size, hipMemcpyDeviceToHost) );
  checkCuda( hipMemcpy(lead3, d_lead3, chunk_size, hipMemcpyDeviceToHost) );
  // Find the index of the max element
  for (i = 0; i < chunk_length; i++) {
    if (lead1[i] && !start1) {
      start1 = i;
    }
    if (lead2[i] && !start2) {
      start2 = i;
    }
    if (lead3[i] && !start3) {
      start3 = i;
    }
    if (start1 && start2 && start3) {
      break;
    }
  }
  minstart = std::min(std::min(start1, start2), start3);
  maxstart = std::max(std::max(start1, start2), start3);
  // Get offsets and prospective new length
  offset1 = start1 - minstart;
  offset2 = start2 - minstart;
  offset3 = start3 - minstart;
  sync_length = lead_length - (maxstart - minstart);

  // Merge
  num_blocks = sync_length / threads_per_block;
  * merged_length_out = num_blocks * threads_per_block;
  // Allocate the output
  checkCuda( hipMalloc((void **) merged_out, * merged_length_out * sizeof(int)) );
  // Merge kernel
  KERNEL(merge_leads)(* merged_out, d_lead1, offset1, d_lead2, offset2, d_lead3, offset3);
  checkCuda( hipDeviceSynchronize() );
  checkCuda( hipGetLastError() );
}

void get_hr(int * out_samples,
            int * out_rr_values,
            int * out_length,
            uint16_t * compressed_lead_1,
            uint16_t * compressed_lead_2,
            uint16_t * compressed_lead_3,
            int lead_length,
            float * d_wavelet,
            int wavelet_length,
            int sampling_rate)
{
  size_t lead_size = lead_length * sizeof(float);
  size_t int_lead_size = lead_length * sizeof(int);
  size_t compressed_lead_size = lead_length * sizeof(uint16_t);
  uint16_t * d_clead1, * d_clead2, * d_clead3;
  float * d_lead1, * d_lead2, * d_lead3;
  float * d_corr1, * d_corr2, * d_corr3;
  int * d_thresh1, * d_thresh2, * d_thresh3;
  int * d_merged;
  int * d_edge;
  int * d_masks;
  int * d_indecies;
  int * d_scan;
  int * d_scatter;
  int * d_rr;
  int * d_filtered;
  int merged_length;
  // Still hardcoded...
  float threshold_value = 0.3;
  int threads_per_block = 200;
  int num_blocks = lead_length / threads_per_block;
  int chunk_length = sampling_rate * 2;
  int reduce_by = 32;
  int reduced_length;
  size_t reduced_size;
  int compacted_length;
  size_t compacted_size;

  // Init
  checkCuda( hipSetDevice(0) );

  // Allocate leads
  // Compressed
  checkCuda( hipMalloc((void **) & d_clead1, compressed_lead_size) );
  checkCuda( hipMalloc((void **) & d_clead2, compressed_lead_size) );
  checkCuda( hipMalloc((void **) & d_clead3, compressed_lead_size) );
  // Decompressed
  checkCuda( hipMalloc((void **) & d_lead1, lead_size) );
  checkCuda( hipMalloc((void **) & d_lead2, lead_size) );
  checkCuda( hipMalloc((void **) & d_lead3, lead_size) );
  // Correlated
  checkCuda( hipMalloc((void **) & d_corr1, lead_size) );
  checkCuda( hipMalloc((void **) & d_corr2, lead_size) );
  checkCuda( hipMalloc((void **) & d_corr3, lead_size) );

  // Transfer leads
  // TODO add streaming
  checkCuda( hipMemcpy(d_clead1, compressed_lead_1, compressed_lead_size, hipMemcpyHostToDevice) );
  checkCuda( hipMemcpy(d_clead2, compressed_lead_2, compressed_lead_size, hipMemcpyHostToDevice) );
  checkCuda( hipMemcpy(d_clead3, compressed_lead_3, compressed_lead_size, hipMemcpyHostToDevice) );

  // Preprocess kernels

  // "Decompress" on GPU (16 bit float to 32 bit float)
  KERNEL(to_float)(d_lead1, (half *) d_clead1, lead_length);
  checkCuda( hipDeviceSynchronize() );
  checkCuda( hipGetLastError() );
  KERNEL(to_float)(d_lead2, (half *) d_clead2, lead_length);
  checkCuda( hipDeviceSynchronize() );
  checkCuda( hipGetLastError() );
  KERNEL(to_float)(d_lead3, (half *) d_clead3, lead_length);
  checkCuda( hipDeviceSynchronize() );
  checkCuda( hipGetLastError() );

  // Free unneeded memory
  hipFree(d_clead1);
  hipFree(d_clead2);
  hipFree(d_clead3);

  // Cross-Correlate with wavelet
  KERNEL(cross_correlate_with_wavelet)(d_corr1, d_lead1, d_wavelet, lead_length, wavelet_length);
  checkCuda( hipDeviceSynchronize() );
  checkCuda( hipGetLastError() );
  KERNEL(cross_correlate_with_wavelet)(d_corr2, d_lead2, d_wavelet, lead_length, wavelet_length);
  checkCuda( hipDeviceSynchronize() );
  checkCuda( hipGetLastError() );
  KERNEL(cross_correlate_with_wavelet)(d_corr3, d_lead3, d_wavelet, lead_length, wavelet_length);
  checkCuda( hipDeviceSynchronize() );
  checkCuda( hipGetLastError() );
  // Free unneeded memory
  hipFree(d_lead1);
  hipFree(d_lead2);
  hipFree(d_lead3);
  hipFree(d_wavelet);

  // Threshold
  // Allocate output
  checkCuda( hipMalloc((void **) & d_thresh1, int_lead_size) );
  checkCuda( hipMalloc((void **) & d_thresh2, int_lead_size) );
  checkCuda( hipMalloc((void **) & d_thresh3, int_lead_size) );
  // Threshold Kernel
  KERNEL(threshold)(d_thresh1, d_corr1, threshold_value);
  checkCuda( hipDeviceSynchronize() );
  checkCuda( hipGetLastError() );
  KERNEL(threshold)(d_thresh2, d_corr2, threshold_value);
  checkCuda( hipDeviceSynchronize() );
  checkCuda( hipGetLastError() );
  KERNEL(threshold)(d_thresh3, d_corr3, threshold_value);
  checkCuda( hipDeviceSynchronize() );
  checkCuda( hipGetLastError() );
  // Free unneeded memory
  hipFree(d_corr1);
  hipFree(d_corr2);
  hipFree(d_corr3);

  // Synchronize and Merge 3 Leads
  // FIXME outputs an array of sparse 2's (not 1's)
  synchronize_and_merge(& d_merged, & merged_length, d_thresh1, d_thresh2, d_thresh3, lead_length, chunk_length);
  // Free unneeded memory
  hipFree(d_thresh1);
  hipFree(d_thresh2);
  hipFree(d_thresh3);

  // Heartrate kernels
  checkCuda( hipMalloc((void **) & d_edge, merged_length * sizeof(int)) );
  reduced_length = merged_length / reduce_by;
  reduced_size = reduced_length * sizeof(int);
  num_blocks = merged_length / threads_per_block;
  KERNEL(edge_detect)(d_edge, d_merged, merged_length);
  checkCuda( hipDeviceSynchronize() );
  checkCuda( hipGetLastError() );
  // Free unneeded memory
  hipFree(d_merged);

  // Pre-Collapse Sparse (32x reduction)
  // Allocate and Zero output
  checkCuda( hipMalloc((void **) & d_masks, reduced_size) );
  checkCuda( hipMalloc((void **) & d_indecies, reduced_size) );
  checkCuda( hipMemset(d_masks, 0, reduced_size) );
  checkCuda( hipMemset(d_indecies, 0, reduced_size) );
  // reduction kernel
  KERNEL(index_of_peak)(d_indecies, d_masks, d_edge);
  checkCuda( hipDeviceSynchronize() );
  checkCuda( hipGetLastError() );
  // Free unneeded memory
  hipFree(d_edge);

  // Collapse Sparse (Stream Compaction)
  threads_per_block = 64;
  num_blocks = (reduced_length / threads_per_block) + 1;
  hipMalloc((void **) & d_scan, reduced_size);
  // Scan
  exclusive_scan(d_scan, d_masks, reduced_length);
  // Get new length
  device_index(d_scan, & compacted_length, reduced_length - 1);
  compacted_size = compacted_length * sizeof(int);
  // Scatter
  num_blocks = (compacted_length / threads_per_block) + 1;
  hipMalloc((void **) & d_scatter, compacted_size);
  KERNEL(scatter)(d_scatter, d_indecies, d_scan, d_masks, compacted_length);
  checkCuda( hipDeviceSynchronize() );
  checkCuda( hipGetLastError() );
  // Free unneeded memory
  hipFree(d_scan);
  hipFree(d_masks);

  // Get heartrate
  hipMalloc((void **) & d_rr, compacted_size);
  KERNEL(get_compact_rr)(d_rr, d_scatter, sampling_rate, compacted_length);
  checkCuda( hipDeviceSynchronize() );
  checkCuda( hipGetLastError() );

  // Remove all values outside the range (40..140) starting at point 1 (i.e. ignore point 0)
  KERNEL(clean_result)(d_rr, 140, 40, 1, compacted_length);
  checkCuda( hipDeviceSynchronize() );
  checkCuda( hipGetLastError() );

  // Moving average filter
  hipMalloc((void **) & d_filtered, compacted_size);
  hipMalloc((void **) & d_scan, compacted_size);
  exclusive_scan(d_scan, d_rr, compacted_length);
  // use a 250 point window for the moving average
  KERNEL(moving_average)(d_filtered, d_scan, 250, compacted_length);
  checkCuda( hipDeviceSynchronize() );
  checkCuda( hipGetLastError() );
  // Free unneeded memory
  hipFree(d_scan);
  hipFree(d_rr);

  // Transfer back to host
  // Copy back
  checkCuda( hipMemcpy(out_samples, d_scatter, compacted_size, hipMemcpyDeviceToHost) );
  checkCuda( hipMemcpy(out_rr_values, d_filtered, compacted_size, hipMemcpyDeviceToHost) );
  // Free unneeded memory
  hipFree(d_indecies);
  hipFree(d_scatter);
  hipFree(d_filtered);
  // Correct first value of output heartrate (it's always wrong)
  out_rr_values[0] = out_rr_values[1];
  // Set the output length
  * out_length = compacted_length;
}

extern "C" {
  void process(int * out_hr,
               int * out_samples,
               int * out_length,
               float * lead1,
               float * lead2,
               float * lead3,
               int lead_length,
               int sampling_rate)
  {
    // Calculate wavelet
    sampling_rate = sampling_rate / 4;
    int wavelet_length = ((int) (0.08 * ((float) sampling_rate))) + 2;
    size_t wavelet_size = wavelet_length * sizeof(float);
    float sigma = 1.0;
    float maxval = 4 * sigma;
    float minval = -maxval;
    float * d_wavelet;
    int num_blocks = 1;
    int threads_per_block = wavelet_length;
    double start, compress, end;

    checkCuda( hipMalloc((void **) & d_wavelet, wavelet_size) );
    KERNEL(mexican_hat)(d_wavelet, sigma, minval, (maxval - minval)/wavelet_length);

    // Compress leads

    int compressed_lead_length = lead_length / 4;
    size_t compressed_lead_size = compressed_lead_length * sizeof(uint16_t);
    uint16_t * compressed_lead1, * compressed_lead2, * compressed_lead3;

    compressed_lead1 = (uint16_t *) malloc(compressed_lead_size);
    assert(compressed_lead1);
    compressed_lead2 = (uint16_t *) malloc(compressed_lead_size);
    assert(compressed_lead2);
    compressed_lead3 = (uint16_t *) malloc(compressed_lead_size);
    assert(compressed_lead3);

    start = get_time();

    // Losing our QRS all of a sudden...
    parallel_turning_point_compress(compressed_lead1, lead1, lead_length);
    parallel_turning_point_compress(compressed_lead2, lead2, lead_length);
    parallel_turning_point_compress(compressed_lead3, lead3, lead_length);

    compress = get_time();

    // Call get_hr

    get_hr(out_hr, out_samples, out_length, compressed_lead1, compressed_lead2, compressed_lead3, compressed_lead_length, d_wavelet, wavelet_length, sampling_rate);

    end = get_time();
    printf("Compression: %lf ms.\n", (compress - start) / 1000.0);
    printf("Total: %lf ms.\n", (end - start) / 1000.0);

    free(compressed_lead1);
    free(compressed_lead2);
    free(compressed_lead3);

  }
}
